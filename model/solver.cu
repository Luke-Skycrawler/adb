#include "cuda_globals.cuh"
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipsolver.h>

hipsolverHandle_t dnHandle;
hipsolverSpHandle_t cusolverSpH;
hipsparseHandle_t cusparseH;
hipStream_t stream;
hipsparseMatDescr_t spdescrA;
csrcholInfo_t sp_chol_info;
// hipblasFillMode_t dnUplo;
// hipblasHandle_t blasHandle;

void setCublasAndCuSparse()
{
	hipsolverDnCreate(&dnHandle);
	// dnUplo = HIPBLAS_FILL_MODE_LOWER;
	// hipblasCreate(&blasHandle);
	hipsolverSpCreate(&cusolverSpH);
	hipStreamCreate(&stream);
	hipsolverSpSetStream(cusolverSpH, stream);
	hipsparseCreate(&cusparseH);
	hipsparseSetStream(cusparseH, stream);
	hipsparseCreateMatDescr(&spdescrA);
	cusolverSpCreateCsrcholInfo(&sp_chol_info);
	hipsparseSetMatType(spdescrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(spdescrA, HIPSPARSE_INDEX_BASE_ZERO);
}

void freeCublasAndCusparse()
{
	hipsolverDnDestroy(dnHandle);
	// hipblasDestroy(blasHandle);
	hipsolverSpDestroy(cusolverSpH);
	hipStreamDestroy(stream);
	hipsparseDestroy(cusparseH);
	cusolverSpDestroyCsrcholInfo(sp_chol_info);
}

void gpuCholSolver(CsrSparseMatrix& hess, float* x)
{
    // hess must be filled by all nonzero value.
    float tol = 1.e-12f;
    const int reorder = 0; // symrcm
    int singularity = 0;


    auto values = thrust::raw_pointer_cast(hess.values.data());
    auto outer_start = thrust::raw_pointer_cast(hess.outer_start.data());
    auto inner = thrust::raw_pointer_cast(hess.inner.data());
    auto rhs = thrust::raw_pointer_cast(host_cuda_globals.b);
    hipsolverStatus_t t = hipsolverSpScsrlsvchol(
        cusolverSpH, hess.rows, hess.nnz,
        spdescrA, values, outer_start, inner,
        rhs, tol, reorder, x, &singularity);
    hipDeviceSynchronize();
    if (0 <= singularity)
    {
        printf("WARNING: the matrix is singular at row %d under tol (%E)\n", singularity, tol);
    }

    //checkNumericalPrecisionHost(m_activeDims, x);
}


