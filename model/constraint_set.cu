#include "hip/hip_runtime.h"
#include "cuda_header.cuh"
#include "cuda_globals.cuh"
#include <thrust/sort.h>
#include <thrust/unique.h>
#include "autogen/autogen.cuh"
using namespace std;

__device__ __host__ float vf_distance(vec3f _v, Facef f, int& _pt_type);
__host__ __device__ void dev_project_to_psd(int dim, float* A){

}

__host__ __device__ i2 offset_and_stride(int I, const i2* lut, int* outers)
{
    // TEST COVERED
    int i = lut[I][0];
    int j = lut[I][1];
    int col_start = outers[i * 12] / (12 * 12);
    int sub_mat_offset = I - col_start;
    int sub_mat_start = outers[i * 12] + sub_mat_offset * 12;
    int stride = outers[i * 12 + 1] - outers[i * 12];

    return { sub_mat_start, stride };
}

__global__ void fill_inner_outers_kernel(int n_cubes, int lut_size, const i2 * lut, int *inners, int *outers) {
    // TESTED
    auto tid = threadIdx.x;
    auto n_task_per_thread = (lut_size+ blockDim.x - 1)/ blockDim.x;  
    // precondition: lut contains all the symmetric pairs (i, j) and (j, i), sorted in ascending order

    for (int _i = 0; _i < n_task_per_thread; _i ++) {
        // detect stairs, fill outers
        auto I = _i + n_task_per_thread * tid;
        if (I < lut_size) {
            int i = lut[I][0];
            int last = I == 0 ? -1: lut[I - 1][0];
            
            if (last != i) {
                inners[i] = I;
                // temporary storage, outers should be filled according to this later
            }
        }
    }
    __syncthreads();
    int n_cols_per_thread = (n_cubes + blockDim.x - 1) / blockDim.x;
    for (int _i = 0; _i < n_cols_per_thread; _i ++) {
        auto I = _i + n_cols_per_thread * tid;
        if (I < n_cubes) {
            auto next = I == n_cubes - 1 ? lut_size: inners[I + 1];
            auto stride = next - inners[I];
            auto start = inners[I] * 12 * 12;

            for (int i =0; i < 12; i ++) {
                outers[I * 12 + i] = start + 12 * (i * stride);
            }
        }
    }
    __syncthreads(); // wait until finish, then break inners array

    int n_block_per_thread = (lut_size + blockDim.x - 1) / blockDim.x;
    for (int _i = 0; _i < n_block_per_thread; _i ++) {
        // fiil inners
        auto I = _i + n_block_per_thread * tid;
        if (I < lut_size) {

            i2 os = offset_and_stride(I, lut, outers);
            // int i = lut[I][0];
            // int j = lut[I][1];
            // int col_start = outers[i * 12] / (12 * 12);
            // int sub_mat_offset = I - col_start;
            // int sub_mat_start = outers[i * 12] + sub_mat_offset * 12;
            // int stride = outers[i * 12 + 1] - outers[i * 12];
            int sub_mat_start = os[0];
            int stride = os[1];
            int j = lut[I][1];
            for (int c = 0; c < 12; c++) {
                for (int r = 0; r < 12; r++) {
                    inners[sub_mat_start + stride * c + r] = j * 12 + r;
                }
            }
        }
    }

}
void build_csr(int n_cubes, const thrust::device_vector<i2> &lut, CsrSparseMatrix & sparse_matrix) {
    // TESTED
    int lut_size = lut.size();
    int nnz = lut_size * 12 * 12;
    sparse_matrix.rows = n_cubes * 12;
    sparse_matrix.cols = n_cubes * 12;
    sparse_matrix.nnz = nnz;

    sparse_matrix.outer_start.resize(n_cubes * 12);
    sparse_matrix.inner.resize(nnz);
    sparse_matrix.values.resize(nnz);
    
    thrust::device_vector<int> dev_inner(nnz), dev_outer(n_cubes * 12);
    
    thrust::fill(sparse_matrix.values.begin(), sparse_matrix.values.end(), 0.0f);
    
    auto lut_ptr = thrust::raw_pointer_cast(lut.data());
    auto inner_ptr = thrust::raw_pointer_cast(dev_inner.data());
    auto outer_ptr = thrust::raw_pointer_cast(dev_outer.data());
    
    fill_inner_outers_kernel<<<1, n_cuda_threads_per_block>>>(n_cubes, lut_size, lut_ptr, inner_ptr, outer_ptr);
    
    sparse_matrix.inner = dev_inner;
    sparse_matrix.outer_start = dev_outer;
    CUDA_CALL(hipGetLastError());
}


__host__ __device__ void pt_grad_hess12x12(vec3f *pt, 
    float *pt_grad, float *pt_hess, bool psd, 
    float *buf  // for local grad and hess return
)
{

    int type;
    auto dist = vf_distance(pt[0], Facef{pt[1], pt[2], pt[3]}, type);
    dev::point_triangle_distance_gradient(pt[0], pt[1], pt[2], pt[3], pt_grad, type, buf);
    dev::point_triangle_distance_hessian(pt[0], pt[1], pt[2], pt[3], pt_hess, type, buf);

    auto B_ = dev::barrier_derivative_d(dist);
    auto B__ = dev::barrier_second_derivative(dist);

    // pt_hess = pt_hess * B_ + pt_grad * pt_grad.transpose() * B__;
    for (int i = 0; i < 12; i++)
       for (int j = 0; j < 12; j++) {
           pt_hess[j * 12 + i] = B_ * pt_hess[j * 12 + i] + pt_grad[i] * pt_grad[j] * B__;
           // column major
       }
    for (int i = 0; i < 12; i++)
       pt_grad[i] *= B_;

    if (psd)
       dev_project_to_psd(12, pt_hess);
}

__host__ __device__ int binary_search(int lut_size, i2* lut, i2 value)
{
    int l = 0, u = lut_size - 1;
    while (l < u) {
        int mid = (l + u) / 2;
        if (lut[mid] < value)
            l = mid + 1;
        else
            u = mid;
    }
    // u is the immediate value >= x
    return u;
}

__forceinline__ int __device__ __host__ rc_to_1d(int r, int c)
{
    return c * 12 + r;
}

// void ipc_term_pt(int npt, i2 *ij, i2 *body_index, int lut_size, i2 * lut)
// __global__ void ipc_pt_batch_kernel(int npt, vec3f *p, Facef * t) {
// }

// __device__ void JTJ(vec3f a, float* ipc_hess)
// {
// }

__device__ void plain_matrix_product(int ar, int ac, int bc, float* a, float* b, float* c)
{
    // assume all in column major
    for (int i = 0; i < ar; i++) {
        for (int j = 0; j < bc; j++) {
            float cij = 0.0f;

            for (int k = 0; k < ac; k++) {
                // cij += aik * bkj
                cij += a[i + k * ar] * b[k + j * ac];
            }
            c[i + j * ar] = cij;
        }
    }
}

__device__ void put(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            atomicAdd(values + offset + c * stride + r, mat12x12[c * 12 + r]);
        }
}


__device__ void put_T(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            atomicAdd(values + offset + c * stride + r, mat12x12[r * 12 + c]);
        }
}
#define CPU_REF
#ifdef CPU_REF
void _put(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            values[offset + c * stride + r]+= mat12x12[c * 12 + r];
        }
}


void _put_T(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            values [offset + c * stride + r]+= mat12x12[r * 12 + c];
        }
}
#endif
__host__ __device__ i2 to_os(i2 ij, int lut_size, i2* lut, int* outers)
{
    int k = binary_search(lut_size, lut, ij);
    return offset_and_stride(k, lut, outers);
}

void get_submat_glue(
    int ii, int jj, 
    float *submat12x12
) 
{
    auto &g{host_cuda_globals};
    auto &h {g.hess};
    auto outers = from_thrust(h.outer_start);
    auto values = from_thrust(h.values);
    auto lut = from_thrust(g.lut);
    auto inners = from_thrust(h.inner);
    auto os = to_os(i2{ii, jj}, g.lut_size, lut.data(), outers.data());
    int offset = os[0], stride = os[1];
    for (int c = 0; c < 12; c++) for (int r = 0; r < 12; r ++){
        submat12x12[c * 12 + r] = values[offset + c * stride + r];
        if (inners[offset + c * stride + r] != jj * 12 + r) {
            printf("index error, should be (%d %d) but inner index = %d\n", jj * 12 + r, ii * 12 + c, inners[offset + c * stride + r]);
        }
    }
}

__global__ void ipc_pt_kernel(
    int npt,
    i2* pt, i2* ij,
    cudaAffineBody* cubes,

    int lut_size, i2* lut,

    float* values, int* outers,
    // CsrSparseMatrix& sparse_hess,
    float* b, // rhs
    float* buffer,
    float* lambdas, float* Tk

)
{

    // input: pt data, body index, is static
    // output: basis Tk (2x12), lambda, gradient g (12x1), hessian H (12x12)

    auto tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto grad_start = buffer;
    auto hess_start = buffer + 12 * blockDim.x;
    auto hess_p_start = hess_start + 144 * blockDim.x;
    auto hess_t_start = hess_p_start + 144 * blockDim.x;
    auto off_diag_start = hess_t_start + 144 * blockDim.x;

    // FIXME: align with cacheline size


    int n_tasks_per_thread = (npt + blockDim.x - 1) / blockDim.x;

    for (int _i = 0; _i < n_tasks_per_thread; _i++) {
        int I = tid * n_tasks_per_thread + _i;
        if (I < npt) {

            // printf("tid = %d  created\n", tid);
            int ii = ij[I][0], jj = ij[I][1];
            auto &ci{ cubes[ii] }, &cj{ cubes[jj] };
            // printf("tid = %d  ci, cj access, ij = {%d, %d}\n pt[I] = {%d, %d}", tid, ii, jj, pt[I][0], pt[I][1]);

            auto fp {cj.triangle(pt[I][1])};
            vec3f projected[4] {
                ci.projected[pt[I][0]],
                fp.t0,
                fp.t1,
                fp.t2
            };
            // printf("tid = %d  projected vertices\n", tid);
            
            float* ipc_hess = hess_start + 144 * tid;
            float* pt_grad = grad_start + tid * 12;

            float *hess_p, *hess_t, *off_diag; 
            float *dgp = ipc_hess, *dgt = ipc_hess + 12;
            // reuse ipc_hess buffer, when dgp computation ipc_hess should be used up

            hess_p = hess_p_start + 144 * tid;
            hess_t = hess_t_start + 144 * tid;
            off_diag = off_diag_start + 144 * tid;

            pt_grad_hess12x12(projected, pt_grad, ipc_hess, true, hess_p);
            const auto all_zero = [](float* a) -> int {
                for (int i = 0; i < 144; i++) {
                    if (a[i] != 0.0f) return 0;
                }
                return 1;
            };

            // printf("tid = %d  pt grad hess checkpoint passed\n", tid);
            // printf("pt grad hess zero status: %d\n", all_zero(ipc_hess));

            vec3f p_tile, t0_tile, t1_tile, t2_tile;
            p_tile = ci.vertices[pt[I][0]];
            Facef f = cj.triangle_at_rest(pt[I][1]);
            t0_tile = f.t0;
            t1_tile = f.t1;
            t2_tile = f.t2;

            float kerp[4]{ 1.0, p_tile.x, p_tile.y, p_tile.z },
                kert[3][4]{
                    { 1.0, t0_tile.x, t0_tile.y, t0_tile.z },
                    { 1.0, t1_tile.x, t1_tile.y, t1_tile.z },
                    { 1.0, t2_tile.x, t2_tile.y, t2_tile.z }
                };


            for (int i = 0; i < 4; i++)
                for (int j = 0; j < 4; j++) {
                    // set hess_p
                    for (int c = 0; c < 3; c++)
                        for (int r = 0; r < 3; r++) {
                            hess_p[rc_to_1d(i * 3 + r, j * 3 + c)] = ipc_hess[rc_to_1d(r, c)] * kerp[i] * kerp[j];
                        }


                    // set hess_t
                    for (int c=  0; c < 3; c++)
                        for (int r= 0; r< 3; r++) {
                            hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                        }
                    for (int k = 0; k < 3; k++)
                        for (int l = 0; l < 3; l++) {
                            for (int c = 0; c < 3; c++)
                                for (int r = 0; r < 3; r++) {
                                    hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d((k + 1) * 3 + r, (l + 1) * 3 + c)] * kert[k][i] * kert[l][j];
                                }
                        }

                    // set off_diag
                    for (int c=  0; c < 3; c++)
                        for (int r= 0; r< 3; r++) {
                            off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                        }
                    for (int l = 0; l < 3; l++) {
                        for (int c = 0; c < 3; c++) {
                            for (int r = 0; r < 3; r++) {
                                off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d(r, (l + 1) * 3 + c)] * (kerp[i] * kert[l][j]);
                            }
                        }
                    }
                }
                
            // printf("tid = %d  grad_p, grad_t checkpoint passed\n", tid);
            // printf(" zero status: p: %d t: %d, off_diag: %d\n", all_zero(hess_p), all_zero(hess_t), all_zero(off_diag));
            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 3; j++) {
                    dgp[i * 3 + j] = pt_grad[j] * kerp[i];
                    dgt[i * 3 + j] = pt_grad[j + 3] * kert[0][i]
                        + pt_grad[j + 6] * kert[1][i]
                        + pt_grad[j + 9] * kert[2][i];
                }
            }
            // printf("tid = %d  dgp checkpoint passed\n", tid);

            auto osii = to_os({ ii, ii }, lut_size, lut, outers);
            auto osij = to_os({ ii, jj }, lut_size, lut, outers);
            auto osjj = to_os({ jj, jj }, lut_size, lut, outers);
            auto osji = to_os({ jj, ii }, lut_size, lut, outers);

            if (ci.mass > 0.0f) {
                if (cj.mass > 0.0f)
                    put_T(values, osji, off_diag);
                put(values, osii, hess_p);
                for (int i = 0; i < 12; i++) {
                    atomicAdd(b + i + ii * 12, dgp[i]);
                }
            }
            if (cj.mass > 0.0f) {
                if (ci.mass > 0.0f)
                    put(values, osij, off_diag);
                put(values, osjj, hess_t);
                for (int i = 0; i < 12; i++) {
                    atomicAdd(b + i + jj * 12, dgt[i]);
                }
            }
            // printf("tid = %d  output checkpoint passed, success\n", tid);

        }
    }
}

__host__ __device__ void ee_grad_hess12x12(vec3f *ee, float *ee_grad, float *ipc_hess, float * buf_start) {
    int type;
    
    float *buf = buf_start;
    float *mollifier_grad = buf;
    buf += 12;
    float* mollifier_hess = buf;
    buf += 144;
    
    type = dev::edge_edge_distance_type(ee[0], ee[1], ee[2], ee[3]);
    float dist = dev::edge_edge_distance(ee[0], ee[1], ee[2], ee[3], type);
    auto ei = ee[1] - ee[0], ej = ee[3] - ee[2];
    float eps_x = dev::eps * dot(ei, ei) * dot(ej, ej);

    float p = dev::edge_edge_mollifier(ee[0], ee[1], ee[2], ee[3], eps_x);
    dev::edge_edge_mollifier_gradient(ee[0], ee[1], ee[2], ee[3], eps_x, mollifier_grad);
    dev::edge_edge_mollifier_hessian(ee[0], ee[1], ee[2], ee[3], eps_x, mollifier_grad, mollifier_hess);
    
    
    
    dev::edge_edge_distance_gradient(ee[0], ee[1], ee[2], ee[3], ee_grad, type, buf);
    dev::edge_edge_distance_hessian(ee[0], ee[1], ee[2], ee[3], ipc_hess, type, buf);

    float B = dev::barrier_function(dist);
    float B_ = dev::barrier_derivative_d(dist);
    float B__ = dev::barrier_second_derivative(dist);

    for (int I = 0; I < 144; I++) {
        int i = I % 12, j = I / 12; // column major
        ipc_hess[i] = mollifier_hess[i] * B + B_ * (mollifier_grad[i] * ee_grad[j] + mollifier_grad[j] * ee_grad[i]) + p * (B__ * ee_grad[i] * ee_grad[j] + B_ * ipc_hess[i]);
    }
    for (int i = 0; i < 12; i++) {
        ee_grad[i] = ee_grad[i] * B_ + mollifier_grad[i] * B;
    }
    dev_project_to_psd(12, ipc_hess);
}

__global__ void put_inertia_kernel(
    int n_cubes, 
    cudaAffineBody *cubes,
    int lut_size, i2 *lut,
    float *values, int *outers,
    float *b,
    float *diag
) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_tasks_per_thread = (n_cubes + blockDim.x - 1) / blockDim.x;
    for (int _i =0; _i < n_task_per_thread; _i ++) {
        int I = _i + tid * n_tasks_per_thread;
        if (I < n_cubes){
            auto osii = to_os({ I, I }, lut_size, lut, outers);
            int offset = osii[0], stride = osii[1];

            for (int c = 0; c < 12; c++)
                for (int r =0; r < 12; r ++) {
                    values[offset + c * stride + r] += diag[c * 12 + r];
                }
            // grad is already added to globals.b
        }
    }
}
__global__ void ipc_ee_kernel(
    int nee, 
    i2 *ee, i2 *ij, 
    cudaAffineBody *cubes,
    int lut_size, i2 *lut,
    float *values, int *outers,
    float *b,
    float *buffer,
    float *lambdas, float *Tk
) {
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto grad_start = buffer;
    auto hess_start = buffer + 12 * blockDim.x;
    auto hess_0_start = hess_start + 144 * blockDim.x;
    auto hess_1_start = hess_0_start + 144 * blockDim.x;
    auto off_diag_start = hess_1_start + 144 * blockDim.x;

    int n_tasks_per_thread = (nee + blockDim.x - 1) / blockDim.x;

    for (int _i = 0; _i < n_tasks_per_thread; _i++) {
        int I = tid * n_tasks_per_thread + _i;
        if (I < nee) {
            int ii = ij[I][0], jj = ij[I][1];
            auto &ci{ cubes[ii] }, &cj{ cubes[jj] };

            auto ei {ci.edge(ee[I][0])}, ej {cj.edge(ee[I][1])};
            vec3f projected[4] {
                ei.e0, ei.e1, ej.e0, ej.e1
            };
            float* ipc_hess = hess_start + 144 * tid;
            float* ee_grad = grad_start + tid * 12;

            float *hess_0, *hess_1, *off_diag; 
            float *dg0 = ipc_hess, *dg1 = ipc_hess + 12;
            hess_0 = hess_0_start + 144 * tid;
            hess_1 = hess_1_start + 144 * tid;
            off_diag = off_diag_start + 144 * tid;

            ee_grad_hess12x12(projected, ee_grad, ipc_hess, hess_0);

            vec3f ei0_tile, ei1_tile, ej0_tile, ej1_tile;
            auto eir {ci.edge_at_rest(ee[I][0])}, ejr {cj.edge_at_rest(ee[I][1])};
            ei0_tile = eir.e0; ei1_tile = eir.e1;
            ej0_tile = ejr.e0; ej1_tile = ejr.e1;

            float ker0[][4] {
                {1.0f, ei0_tile.x, ei0_tile.y, ei0_tile.z},
                {1.0f, ei1_tile.x, ei1_tile.y, ei1_tile.z},
            }, ker1[][4]{
                {1.0f, ej0_tile.x, ej0_tile.y, ej0_tile.z},
                {1.0f, ej1_tile.x, ej1_tile.y, ej1_tile.z}
            };

            // fill hess_0, hess_2, off_diag
            for(int i = 0;  i < 4; i ++)
                for (int j = 0; j < 4; j ++) {
                    for (int c = 0; c < 3; c++)
                        for (int  r = 0; r < 3; r++) {
                            hess_1[rc_to_1d(i * 3+ r, j * 3 + c)] = 0.0f;
                            hess_0[rc_to_1d(i * 3+ r, j * 3 + c)] = 0.0f;
                            off_diag[rc_to_1d(i * 3+ r, j * 3 + c)] = 0.0f;
                        }
                    for (int k = 0; k < 2; k++) 
                        for(int l = 0; l < 2; l ++){
                            
                            for (int c = 0; c < 3; c++)
                                for (int r = 0; r < 3; r++) {
                                    hess_0[rc_to_1d(i *  3 + r, j * 3 + c)] += ipc_hess[rc_to_1d(k * 3 + r, l * 3 + c)] * ker0[k][i] * ker0[l][j];

                                    hess_1[rc_to_1d(i  * 3+ r, j * 3 + j)] += ipc_hess[rc_to_1d((k + 2) * 3 + r, (l + 2) * 3 + c)] * ker1[k][i] * ker1[l][j];

                                    off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d(k * 3 + r, (l + 2) * 3 + c)] * ker0[k][i] * ker1[l][j];
                                }
                        }

                }
            
            // fill dg0, dg1
            for (int i = 0; i < 12; i++) {
                dg0[i] = ker0[0][i / 3] * ee_grad[i % 3] + ker0[1][i / 3] * ee_grad[i % 3 + 3];
                dg1[i] = ker1[0][i / 3] * ee_grad[i % 3 + 6] + ker1[1][i / 3] * ee_grad[i % 3 + 9];
            }
            
            auto osii = to_os({ ii, ii }, lut_size, lut, outers);
            auto osij = to_os({ ii, jj }, lut_size, lut, outers);
            auto osjj = to_os({ jj, jj }, lut_size, lut, outers);
            auto osji = to_os({ jj, ii }, lut_size, lut, outers);

            if (ci.mass > 0.0f) {
                if (cj.mass > 0.0f)
                    put_T(values, osji, off_diag);
                put(values,osii, hess_0);
                for (int i = 0; i < 12; i ++) {
                    atomicAdd(b + i + ii * 12, dg0[i]);
                }
            }
            if (cj.mass > 0.0f) {
                if (ci.mass > 0.0f)
                    put(values, osij, off_diag);
                put(values, osjj, hess_1);
                for (int i = 0; i < 12; i++) {
                    atomicAdd(b + i + jj * 12, dg1[i]);
                }
            }

        }
    }
}
#define CPU_REF
#ifdef CPU_REF
void ipc_pt_cpu(
    int npt,
    i2* pt, i2* ij,
    cudaAffineBody* cubes,

    int lut_size, i2* lut,

    float* values, int* outers,
    // CsrSparseMatrix& sparse_hess,
    float* b, // rhs
    float* buffer,
    float* lambdas, float* Tk

)
{

    // input: pt data, body index, is static
    // output: basis Tk (2x12), lambda, gradient g (12x1), hessian H (12x12)

    // FIXME: align with cacheline size
    // FIXME: make sure ipc autogen is row-major

    for (int I = 0; I < npt; I++) {

        int ii = ij[I][0], jj = ij[I][1];
        auto &ci{ cubes[ii] }, &cj{ cubes[jj] };

        auto fp{ cj.triangle(pt[I][1]) };
        vec3f projected[4]{
            ci.projected[pt[I][0]],
            fp.t0,
            fp.t1,
            fp.t2
        };

        float ipc_hess[144];
        float pt_grad[12];

        float hess_p[144], hess_t[144], off_diag[144];
        float *dgp = ipc_hess, *dgt = ipc_hess + 12;
        // reuse ipc_hess buffer, when dgp computation ipc_hess should be used up

        pt_grad_hess12x12(projected, pt_grad, ipc_hess, true, hess_p);

        vec3f p_tile, t0_tile, t1_tile, t2_tile;
        p_tile = ci.vertices[pt[I][0]];
        Facef f = cj.triangle_at_rest(pt[I][1]);
        t0_tile = f.t0;
        t1_tile = f.t1;
        t2_tile = f.t2;

        float kerp[4]{ 1.0, p_tile.x, p_tile.y, p_tile.z },
            kert[3][4]{
                { 1.0, t0_tile.x, t0_tile.y, t0_tile.z },
                { 1.0, t1_tile.x, t1_tile.y, t1_tile.z },
                { 1.0, t2_tile.x, t2_tile.y, t2_tile.z }
            };

        for (int i = 0; i < 4; i++)
            for (int j = 0; j < 4; j++) {
                // set hess_p
                for (int c = 0; c < 3; c++)
                    for (int r = 0; r < 3; r++) {
                        hess_p[rc_to_1d(i * 3 + r, j * 3 + c)] = ipc_hess[rc_to_1d(r, c)] * kerp[i] * kerp[j];
                    }

                // set hess_t
                for (int c = 0; c < 3; c++)
                    for (int r = 0; r < 3; r++) {
                        hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                    }
                for (int k = 0; k < 3; k++)
                    for (int l = 0; l < 3; l++) {
                        for (int c = 0; c < 3; c++)
                            for (int r = 0; r < 3; r++) {
                                hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d((k + 1) * 3 + r, (l + 1) * 3 + c)] * kert[k][i] * kert[l][j];
                            }
                    }

                // set off_diag
                for (int c = 0; c < 3; c++)
                    for (int r = 0; r < 3; r++) {
                        off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                    }
                for (int l = 0; l < 3; l++) {
                    for (int c = 0; c < 3; c++) {
                        for (int r = 0; r < 3; r++) {
                            off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d(r, (l + 1) * 3 + c)] * (kerp[i] * kert[l][j]);
                        }
                    }
                }
            }

        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 3; j++) {
                dgp[i * 3 + j] = pt_grad[j] * kerp[i];
                dgt[i * 3 + j] = pt_grad[j + 3] * kert[0][i]
                    + pt_grad[j + 6] * kert[1][i]
                    + pt_grad[j + 9] * kert[2][i];
            }
        }

        auto osii = to_os({ ii, ii }, lut_size, lut, outers);
        auto osij = to_os({ ii, jj }, lut_size, lut, outers);
        auto osjj = to_os({ jj, jj }, lut_size, lut, outers);
        auto osji = to_os({ jj, ii }, lut_size, lut, outers);

        if (ci.mass > 0.0f) {
            if (cj.mass > 0.0f)
                _put_T(values, osji, off_diag);
            _put(values, osii, hess_p);
            for (int i = 0; i < 12; i++) {
                b [i + ii * 12] += dgp[i];
            }
        }
        if (cj.mass > 0.0f) {
            if (ci.mass > 0.0f)
                _put(values, osij, off_diag);
            _put(values, osjj, hess_t);
            for (int i = 0; i < 12; i++) {
                b[i + jj * 12] += dgt[i];
            }
        }
    }
}

#endif
void project_glue(int vtn);
void cuda_ipc_glue()
{
    auto& g{ host_cuda_globals };
    auto& lt{ g.leader_thread_buffer_back };
    auto lt_stashed = lt;
    
    auto ps = from_thrust(thrust::device_vector<i2>(g.pt.p, g.pt.p + g.npt));
    auto bs = from_thrust(thrust::device_vector<i2>(g.pt.b, g.pt.b + g.npt));
    

    project_glue(1);
    if (g.params["ipc_cpu_debug"]) {
        float b[12], buf[144];
        auto host_cubes = host_cuda_globals.host_cubes;
        vec3f* host_projected = new vec3f[host_cuda_globals.n_vertices], *host_vertices = new vec3f[host_cuda_globals.n_vertices];
        int* host_edges = new int[host_cuda_globals.n_edges * 2];
        int* host_faces = new int[host_cuda_globals.n_faces * 3];

        int start = 0;
        hipMemcpy(host_projected, host_cuda_globals.projected_vertices, sizeof(vec3f) * host_cuda_globals.n_vertices, hipMemcpyDeviceToHost);
        hipMemcpy(host_edges, host_cuda_globals.edges, sizeof(int) * host_cuda_globals.n_edges * 2, hipMemcpyDeviceToHost);
        hipMemcpy(host_faces, host_cuda_globals.faces, sizeof(int) * host_cuda_globals.n_faces * 3, hipMemcpyDeviceToHost);
        hipMemcpy(host_vertices, host_cuda_globals.vertices_at_rest, sizeof(vec3f) * host_cuda_globals.n_vertices, hipMemcpyDeviceToHost);
        for (int i = 0; i < host_cubes.size(); i++) {
            host_cubes[i].projected = host_projected + start;
            host_cubes[i].vertices  = host_vertices + start;
            start += host_cubes[i].n_vertices;
            host_cubes[i].edges = host_cubes[i].edges - host_cuda_globals.edges + host_edges;
            host_cubes[i].faces = host_cubes[i].faces - host_cuda_globals.faces + host_faces;
        }

        thrust::host_vector<float> ret_values = g.hess.values;
        if (g.params["pt_enable"])
        ipc_pt_cpu(g.npt, ps.data(), bs.data(),
            host_cubes.data(),
            g.lut_size, from_thrust(g.lut).data(),
            ret_values.data(), from_thrust(g.hess.outer_start).data(),
            b, buf,
            nullptr, nullptr);
        // if (g.params["ee_enable"])
        // ipc_ee_cpu(g.nee, from_thrust)

        g.hess.values = ret_values;

        delete []host_projected;
        delete []host_edges;
        delete []host_faces;
        delete []host_vertices;

    }
    else {
        if (g.params["pt_enable"])
        ipc_pt_kernel<<<1, 1>>>(g.npt, g.pt.p, g.pt.b,
            g.cubes,
            g.lut_size, PTR(g.lut),
            PTR(g.hess.values), PTR(g.hess.outer_start),
            g.b, (float*)lt,
            nullptr, nullptr);
        if (g.params["ee_enable"]) 
        ipc_ee_kernel<<<1, 1>>>(g.nee, g.ee.p, g.ee.b,
            g.cubes,
            g.lut_size, PTR(g.lut),
            PTR(g.hess.values), PTR(g.hess.outer_start),
            g.b, (float*)lt,
            nullptr, nullptr);
        put_inertia_kernel<<<1, 1>>>(
            g.n_cubes,
            g.cubes,
            g.lut_size, PTR(g.lut),
            PTR(g.hess.values), PTR(g.hess.outer_start),
            g.b, g.hess_diag
        );
    }
    
    CUDA_CALL(hipDeviceSynchronize());
    const auto all_zero = [](vector<float> values, int n) {
        for (int i = 0; i < n; i++) {
            if (values[i] != 0.0f) return false;
        }
        return true;
    };
    if (g.npt && all_zero(from_thrust(g.hess.values), g.hess.nnz)) {
    printf("\nerror: all zero in sparse matrix\n");
    }
    lt = lt_stashed;
}


inline __host__ __device__ vec3f linerp(vec3f pt1, vec3f pt0, float t){
    return t * pt1 + (1.0f - t) * pt0;
}
inline __host__ __device__ Facef linerp(Facef tt1, Facef tt0, float t) {
    return Facef {
        linerp(tt1.t0, tt0.t0, t),
        linerp(tt1.t1, tt0.t1, t),
        linerp(tt1.t2, tt0.t2, t),
    };
}
inline __host__ __device__ Edgef linerp(Edgef et1, Edgef et0, float t) {
    return Edgef {
        linerp(et1.e0, et0.e0, t),
        linerp(et1.e1, et0.e1, t),
    };
}

__host__ __device__ void cubic_binomial (float a[3], float b[3], float ret_polynomial[4]){
    polynomial[0] += b[0] * b[1] * b[2];
    polynomial[1] += a[0] * b[1] * b[2] + b[0] * b[1] * a[2] + b[0] * a[1] * b[2];
    polynomial[2] += a[0] * a[1] * b[2] + b[0] * a[1] * a[2] + a[0] * b[1] * a[2];
    polynomial[3] += a[0] * a[1] * a[2];
}

__forceinline__ int rc_3(int i, int j) {
    return 3 * j + i;
}
__host__ __device__ void det_polynomial(
    // const mat3& a, const mat3& b
    float *a, float *b, float *ret_polynomial
)
{
    float pos_polynomial[4]{ 0.0 }, neg_polynomial[4]{ 0.0 };
    float c11c22c33[2][3]{
        { a[rc_3(0, 0)], a[rc_3(1, 1)], a[rc_3(2, 2)] },
        { b[rc_3(0, 0)], b[rc_3(1, 1)], b[rc_3(2, 2)] }
    },
        c12c23c31[2][3]{
            { a[rc_3(0, 1)], a[rc_3(1, 2)], a[rc_3(2, 0)] },
            { b[rc_3(0, 1)], b[rc_3(1, 2)], b[rc_3(2, 0)] }
        },
        c13c21c32[2][3]{
            { a[rc_3(0, 2)], a[rc_3(1, 0)], a[rc_3(2, 1)] },
            { b[rc_3(0, 2)], b[rc_3(1, 0)], b[rc_3(2, 1)] }
        };
    float c11c23c32[2][3]{
        { a[rc_3(0, 0)], a[rc_3(1, 2)], a[rc_3(2, 1)] }, { b[rc_3(0, 0)], b[rc_3(1, 2)], b[rc_3(2, 1)] }
    },
        c12c21c33[2][3]{
            { a[rc_3(0, 1)], a[rc_3(1, 0)], a[rc_3(2, 2)] }, { b[rc_3(0, 1)], b[rc_3(1, 0)], b[rc_3(2, 2)] }
        },
        c13c22c31[2][3]{
            { a[rc_3(0, 2)], a[rc_3(1, 1)], a[rc_3(2, 0)] }, { b[rc_3(0, 2)], b[rc_3(1, 1)], b[rc_3(2, 0)] }
        };
    cubic_binomial(
        c11c22c33[0],
        c11c22c33[1],
        pos_polynomial);
    cubic_binomial(
        c12c23c31[0],
        c12c23c31[1],
        pos_polynomial);
    cubic_binomial(
        c13c21c32[0],
        c13c21c32[1],
        pos_polynomial);
    cubic_binomial(
        c11c23c32[0],
        c11c23c32[1],
        neg_polynomial);
    cubic_binomial(
        c12c21c33[0],
        c12c21c33[1],
        neg_polynomial);
    cubic_binomial(
        c13c22c31[0],
        c13c22c31[1],
        neg_polynomial);
    for (int i = 0; i < 4; i++) ret_polynomial[i] = pos_polynomial[i] - neg_polynomial[i];
}



int build_and_solve_4_points_coplanar(
    const vec3f& p0_t0,
    const vec3f& p1_t0,
    const vec3f& p2_t0,
    const vec3f& p3_t0,

    const vec3f& p0_t1,
    const vec3f& p1_t1,
    const vec3f& p2_t1,
    const vec3f& p3_t1,

    float roots[3])
{
    mat3 a1, a2, a3, a4;
    mat3 b1, b2, b3, b4;

    b1 << p1_t0, p2_t0, p3_t0;
    b2 << p0_t0, p2_t0, p3_t0;
    b3 << p0_t0, p1_t0, p3_t0;
    b4 << p0_t0, p1_t0, p2_t0;

    a1 << p1_t1, p2_t1, p3_t1;
    a2 << p0_t1, p2_t1, p3_t1;
    a3 << p0_t1, p1_t1, p3_t1;
    a4 << p0_t1, p1_t1, p2_t1;

    a1 -= b1;
    a2 -= b2;
    a3 -= b3;
    a4 -= b4;

    float a1[9] {
        pt_t1.x - p1_t0.x, pt_t1.y - p1_t0.y, pt_t1.z - p1_t0.z,
        pt_t2.x - p2_t0.x, pt_t2.y - p2_t0.y, pt_t2.z - p2_t0.z,
        pt_t3.x - p3_t0.x, pt_t3.y - p3_t0.y, pt_t3.z - p3_t0.z
    },
    a2[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p2_t1.x - p2_t0.x, p2_t1.y - p2_t0.y, p2_t1.z - p2_t0.z,
        p3_t1.x - p3_t0.x, p3_t1.y - p3_t0.y, p3_t1.z - p3_t0.z
    }, 
    a3[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p1_t1.x - p1_t0.x, p1_t1.y - p1_t0.y, p1_t1.z - p1_t0.z,
        p3_t1.x - p3_t0.x, p3_t1.y - p3_t0.y, p3_t1.z - p3_t0.z
    },
    a4[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p1_t1.x - p1_t0.x, p1_t1.y - p1_t0.y, p1_t1.z - p1_t0.z,
        p2_t1.x - p2_t0.x, p2_t1.y - p2_t0.y, p2_t1.z - p2_t0.z
    };
    float b1[9] {
        p1_t0.x, p1_t0.y, p1_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    }, 
    b2[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    },
    b3[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p1_t0.x, p1_t0.y, p1_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    },
    b4[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p1_t0.x, p1_t0.y, p1_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z
    }; 
    
    float ret_polynomial[4] {0.0f};
    float tmp_polynomial[4] {0.0f};

    det_polynomial(a1, b1, ret_polynomial);
    det_polynomial(a2, b2, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] -= tmp_polynomial[i];
    det_polynomial(a3, b3, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] += tmp_polynomial[i];
    det_polynomial(a4, b4, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] -= tmp_polynomial[i];

    double root = 1.0;
    int found = cubic_roots(roots, ret_polynomial, 0.0, 1.0);
    return found;
}


__device__ __host__ bool _cross(const Edgef &ei, const Edgef &ej){
    auto vei = ei.e1 - ei.e0;
    auto vej0 = ej.e0 - ei.e0;
    auto vej1 = ej.e1 - ei.e0;
    return (cross(vei, vej0), cross(vei, vej1)) < 0.0f;
}

__device__ __host__ bool verify_root_ee(
    const Edgef &ei, 
    const Edgef &ej
) {
    return _cross(ei, ej) && _cross(ej, ei);
}

__forceinline__ __device__ __host__ bool inside(const Facef &f, const vec3f &p) {
    auto f01 = cross(t0 - p, t1- p);
    auto f12 = cross(t1 - p, t2- p);
    auto f20 = cross(t2 - p, t0- p);
    return dot(f01, f12) >= 0.0f && dot(f12, f20) >= 0.0f;
}

__device__ __host__ verify_root_pt(
    const vec3f &p, const Facef &f
 ) {
    auto n = f.unit_normal();
    double d = dot(n, p - f.t0);
    auto v = p - d * n;
    return inside(f, v);
 }
__device__ __host__ float pt_collision_time(
    const vec3f &p0,
    const Facef &t0, 
    const vec3f &p1,
    const Facef &t1
){
    float roots[3];
    int found = build_and_solve_4_points_coplanar(p0, t0.t0, t0.t1, t0.t2, p1, t1.t0, t1.t1, t1.t2, roots);
    bool true_root = false;
    for (int i = 0; i < found && !true_root; i ++) {
        root = roots[i];
        true_root = verify_root_pt(linerp(p1, p0, root), linerp(t1, t0, root));
    }
    return found && true_root? root: 1.0f;
}

__device__ __host__ float ee_collision_time(
    const Edgef &ei0, 
    const Edgef &ej0,
    const Edgef &ei1,
    const Edgef &ej1
){
    float roots[3];
    int found = build_and_solve_4_points_coplanar(
        ei0.e0, ei0.e1, ej0.e0, ej0.e1,
        ei1.e0, ei1.e1, ej1.e0, ej1.e1,
        roots
    );
    float root = 1.0f;
    bool true_root = false;
    for (int i = 0; i < found && !true_root; i ++) {
        root = roots[i];
        true_root = verify_root_ee(linerp(ei1, ei0, root), lerp(ej1, ej0, root));
    }
    return found && true_root? root: 1.0f;
}