#include "hip/hip_runtime.h"
#include "cuda_header.cuh"
#include "cuda_globals.cuh"
#include <thrust/sort.h>
#include <thrust/unique.h>
#include "autogen/autogen.cuh"
using namespace std;

__device__ __host__ float vf_distance(vec3f _v, Facef f, int& _pt_type);
__host__ __device__ void dev_project_to_psd(int dim, float* A){

}

__host__ __device__ i2 offset_and_stride(int I, const i2* lut, int* outers)
{
    // TEST COVERED
    int i = lut[I][0];
    int j = lut[I][1];
    int col_start = outers[i * 12] / (12 * 12);
    int sub_mat_offset = I - col_start;
    int sub_mat_start = outers[i * 12] + sub_mat_offset * 12;
    int stride = outers[i * 12 + 1] - outers[i * 12];

    return { sub_mat_start, stride };
}

__global__ void fill_inner_outers_kernel(int n_cubes, int lut_size, const i2 * lut, int *inners, int *outers) {
    // TESTED
    auto tid = threadIdx.x;
    auto n_task_per_thread = (lut_size+ blockDim.x - 1)/ blockDim.x;  
    // precondition: lut contains all the symmetric pairs (i, j) and (j, i), sorted in ascending order

    for (int _i = 0; _i < n_task_per_thread; _i ++) {
        // detect stairs, fill outers
        auto I = _i + n_task_per_thread * tid;
        if (I < lut_size) {
            int i = lut[I][0];
            int last = I == 0 ? -1: lut[I - 1][0];
            
            if (last != i) {
                inners[i] = I;
                // temporary storage, outers should be filled according to this later
            }
        }
    }
    __syncthreads();
    int n_cols_per_thread = (n_cubes + blockDim.x - 1) / blockDim.x;
    for (int _i = 0; _i < n_cols_per_thread; _i ++) {
        auto I = _i + n_cols_per_thread * tid;
        if (I < n_cubes) {
            auto next = I == n_cubes - 1 ? lut_size: inners[I + 1];
            auto stride = next - inners[I];
            auto start = inners[I] * 12 * 12;

            for (int i =0; i < 12; i ++) {
                outers[I * 12 + i] = start + 12 * (i * stride);
            }
        }
    }
    __syncthreads(); // wait until finish, then break inners array

    int n_block_per_thread = (lut_size + blockDim.x - 1) / blockDim.x;
    for (int _i = 0; _i < n_block_per_thread; _i ++) {
        // fiil inners
        auto I = _i + n_block_per_thread * tid;
        if (I < lut_size) {

            i2 os = offset_and_stride(I, lut, outers);
            // int i = lut[I][0];
            // int j = lut[I][1];
            // int col_start = outers[i * 12] / (12 * 12);
            // int sub_mat_offset = I - col_start;
            // int sub_mat_start = outers[i * 12] + sub_mat_offset * 12;
            // int stride = outers[i * 12 + 1] - outers[i * 12];
            int sub_mat_start = os[0];
            int stride = os[1];
            int j = lut[I][1];
            for (int c = 0; c < 12; c++) {
                for (int r = 0; r < 12; r++) {
                    inners[sub_mat_start + stride * c + r] = j * 12 + r;
                }
            }
        }
    }

}
void build_csr(int n_cubes, const thrust::device_vector<i2> &lut, CsrSparseMatrix & sparse_matrix) {
    // TESTED
    int lut_size = lut.size();
    int nnz = lut_size * 12 * 12;
    sparse_matrix.rows = n_cubes * 12;
    sparse_matrix.cols = n_cubes * 12;
    sparse_matrix.nnz = nnz;

    sparse_matrix.outer_start.resize(n_cubes * 12);
    sparse_matrix.inner.resize(nnz);
    sparse_matrix.values.resize(nnz);
    
    thrust::device_vector<int> dev_inner(nnz), dev_outer(n_cubes * 12);
    
    thrust::fill(sparse_matrix.values.begin(), sparse_matrix.values.end(), 0.0f);
    
    auto lut_ptr = thrust::raw_pointer_cast(lut.data());
    auto inner_ptr = thrust::raw_pointer_cast(dev_inner.data());
    auto outer_ptr = thrust::raw_pointer_cast(dev_outer.data());
    
    fill_inner_outers_kernel<<<1, n_cuda_threads_per_block>>>(n_cubes, lut_size, lut_ptr, inner_ptr, outer_ptr);
    
    sparse_matrix.inner = dev_inner;
    sparse_matrix.outer_start = dev_outer;
    CUDA_CALL(hipGetLastError());
}


__host__ __device__ void pt_grad_hess12x12(vec3f *pt, 
    float *pt_grad, float *pt_hess, bool psd, 
    float *buf  // for local grad and hess return
)
{

    int type;
    auto dist = vf_distance(pt[0], Facef{pt[1], pt[2], pt[3]}, type);
    dev::point_triangle_distance_gradient(pt[0], pt[1], pt[2], pt[3], pt_grad, type, buf);
    dev::point_triangle_distance_hessian(pt[0], pt[1], pt[2], pt[3], pt_hess, type, buf);
    printf("pt grad hess generated\n");

    auto B_ = dev::barrier_derivative_d(dist);
    auto B__ = dev::barrier_second_derivative(dist);

    // pt_hess = pt_hess * B_ + pt_grad * pt_grad.transpose() * B__;
    for (int i = 0; i < 12; i++)
       for (int j = 0; j < 12; j++) {
           pt_hess[j * 12 + i] = B_ * pt_hess[j * 12 + i] + pt_grad[i] * pt_grad[j] * B__;
           // column major
       }
    for (int i = 0; i < 12; i++)
       pt_grad[i] *= B_;
    printf("pt grad, ipc hess processed\n");

    if (psd)
       dev_project_to_psd(12, pt_hess);
    printf("sub program returned\n");
}

__host__ __device__ int binary_search(int lut_size, i2* lut, i2 value)
{
    int l = 0, u = lut_size - 1;
    while (l < u) {
        int mid = (l + u) / 2;
        if (lut[mid] < value)
            l = mid + 1;
        else
            u = mid;
    }
    // u is the immediate value over x
    return u;
}

__forceinline__ int __device__ rc_to_1d(int r, int c)
{
    return c * 12 + r;
}

// void ipc_term_pt(int npt, i2 *ij, i2 *body_index, int lut_size, i2 * lut)
// __global__ void ipc_pt_batch_kernel(int npt, vec3f *p, Facef * t) {
// }

__device__ void JTJ(vec3f a, float* ipc_hess)
{
}

__device__ void plain_matrix_product(int ar, int ac, int bc, float* a, float* b, float* c)
{
    // assume all in column major
    for (int i = 0; i < ar; i++) {
        for (int j = 0; j < bc; j++) {
            float cij = 0.0f;

            for (int k = 0; k < ac; k++) {
                // cij += aik * bkj
                cij += a[i + k * ar] * b[k + j * ac];
            }
            c[i + j * ar] = cij;
        }
    }
}

__device__ void put(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            atomicAdd(values + offset + c * stride + r, mat12x12[c * 12 + r]);
        }
}


__device__ void put_T(float* values, i2 offset_stride, float* mat12x12)
{
    int offset = offset_stride[0], stride = offset_stride[1];
    for (int c = 0; c < 12; c++)
        for (int r = 0; r < 12; r++) {
            atomicAdd(values + offset + c * stride + r, mat12x12[r * 12 + c]);
        }
}

__host__ __device__ i2 to_os(i2 ij, int lut_size, i2* lut, int* outers)
{
    int k = binary_search(lut_size, lut, ij);
    return offset_and_stride(k, lut, outers);
}

void get_submat_glue(
    int ii, int jj, 
    float *submat12x12
) 
{
    auto &g{host_cuda_globals};
    auto &h {g.hess};
    auto outers = from_thrust(h.outer_start);
    auto values = from_thrust(h.values);
    auto lut = from_thrust(g.lut);
    auto os = to_os(i2{ii, jj}, g.lut_size, lut.data(), outers.data());
    int offset = os[0], stride = os[1];
    for (int c = 0; c < 12; c++) for (int r = 0; r < 12; r ++){
        submat12x12[c * 12 + r] = values[offset + c * stride + r];
    }
}

__global__ void ipc_pt_kernel(
    int npt,
    i2* pt, i2* ij,
    cudaAffineBody* cubes,

    int lut_size, i2* lut,

    float* values, int* inners, int* outers,
    // CsrSparseMatrix& sparse_hess,
    float* b, // rhs
    float* buffer,
    float* lambdas, float* Tk

)
{

    // input: pt data, body index, is static
    // output: basis Tk (2x12), lambda, gradient g (12x1), hessian H (12x12)

    // __shared__ float pt_grad_hess[13 * 12 * n_cuda_threads_per_block];
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto grad_start = buffer;
    auto hess_start = buffer + 12 * blockDim.x;
    auto hess_p_start = hess_start + 144 * blockDim.x;
    auto hess_t_start = hess_p_start + 144 * blockDim.x;
    auto off_diag_start = hess_t_start + 144 * blockDim.x;

    // FIXME: align with cacheline size
    // FIXME: make sure ipc autogen is row-major


    int n_tasks_per_thread = (npt + n_cuda_threads_per_block - 1) / n_cuda_threads_per_block;

    for (int _i = 0; _i < n_tasks_per_thread; _i++) {
        int I = tid * n_tasks_per_thread + _i;
        if (I < npt) {

            printf("tid = %d  created\n", tid);
            int ii = ij[I][0], jj = ij[I][1];
            auto &ci{ cubes[ii] }, &cj{ cubes[jj] };
            printf("tid = %d  ci, cj access, ij = {%d, %d}\n pt[I] = {%d, %d}", tid, ii, jj, pt[I][0], pt[I][1]);

            auto fp {cj.triangle(pt[I][1])};
            vec3f projected[4] {
                ci.projected[pt[I][0]],
                fp.t0,
                fp.t1,
                fp.t2
            };
            printf("tid = %d  projected vertices\n", tid);
            
            float* ipc_hess = hess_start + 144 * tid;
            float* pt_grad = grad_start + tid * 12;

            float *hess_p, *hess_t, *off_diag; 
            float *dgp = ipc_hess, *dgt = ipc_hess + 12;
            // reuse ipc_hess buffer, when dgp computation ipc_hess should be used up

            hess_p = hess_p_start + 144 * tid;
            hess_t = hess_t_start + 144 * tid;
            off_diag = off_diag_start + 144 * tid;

            pt_grad_hess12x12(projected, pt_grad, ipc_hess, true, hess_p);
            printf("tid = %d  pt grad hess checkpoint passed\n", tid);

            vec3f p_tile, t0_tile, t1_tile, t2_tile;
            p_tile = ci.vertices[pt[I][0]];
            Facef f = cj.triangle_at_rest(pt[I][1]);
            t0_tile = f.t0;
            t1_tile = f.t1;
            t2_tile = f.t2;

            float kerp[4]{ 1.0, p_tile.x, p_tile.y, p_tile.z },
                kert[3][4]{
                    { 1.0, t0_tile.x, t0_tile.y, t0_tile.z },
                    { 1.0, t1_tile.x, t1_tile.y, t1_tile.z },
                    { 1.0, t2_tile.x, t2_tile.y, t2_tile.z }
                };


            for (int i = 0; i < 4; i++)
                for (int j = 0; j < 4; j++) {
                    // set hess_p
                    for (int c = 0; c < 3; c++)
                        for (int r = 0; r < 3; r++) {
                            hess_p[rc_to_1d(i * 3 + r, j * 3 + c)] = ipc_hess[rc_to_1d(r, c)] * kerp[i] * kerp[j];
                        }


                    // set hess_t
                    for (int c=  0; c < 3; c++)
                        for (int r= 0; r< 3; r++) {
                            hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                        }
                    for (int k = 0; k < 3; k++)
                        for (int l = 0; l < 3; l++) {
                            for (int c = 0; c < 3; c++)
                                for (int r = 0; r < 3; r++) {
                                    hess_t[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d((k + 1) * 3 + r, (l + 1) * 3 + c)] * kert[k][i] * kert[l][j];
                                }
                        }

                    // set off_diag
                    for (int c=  0; c < 3; c++)
                        for (int r= 0; r< 3; r++) {
                            off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] = 0.0f;
                        }
                    for (int l = 0; l < 3; l++) {
                        for (int c = 0; c < 3; c++) {
                            for (int r = 0; r < 3; r++) {
                                off_diag[rc_to_1d(i * 3 + r, j * 3 + c)] += ipc_hess[rc_to_1d(r, (l + 1) * 3 + c)] * (kerp[i] * kert[l][j]);
                            }
                        }
                    }
                }
                
            printf("tid = %d  grad_p, grad_t checkpoint passed\n", tid);

            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 3; j++) {
                    dgp[i * 3 + j] = pt_grad[j] * kerp[i];
                    dgt[i * 3 + j] = pt_grad[j + 3] * kert[0][i]
                        + pt_grad[j + 6] * kert[1][i]
                        + pt_grad[j + 9] * kert[2][i];
                }
            }
            printf("tid = %d  dgp checkpoint passed\n", tid);

            auto osii = to_os({ ii, ii }, lut_size, lut, outers);
            auto osij = to_os({ ii, jj }, lut_size, lut, outers);
            auto osjj = to_os({ jj, jj }, lut_size, lut, outers);
            auto osji = to_os({ jj, ii }, lut_size, lut, outers);

            if (ci.mass > 0.0f) {
                if (cj.mass > 0.0f)
                    put_T(values, osji, off_diag);
                put(values, osii, hess_p);
                for (int i = 0; i < 12; i++) {
                    atomicAdd(b + i + ii * 12, dgp[i]);
                }
            }
            if (cj.mass > 0.0f) {
                if (ci.mass > 0.0f)
                    put(values, osij, off_diag);
                put(values, osjj, hess_t);
                for (int i = 0; i < 12; i++) {
                    atomicAdd(b + i + jj * 12, dgt[i]);
                }
            }
            printf("tid = %d  output checkpoint passed, success\n", tid);

        }
    }
}
void project_glue(int vtn);
void cuda_ipc_glue()
{
    auto& g{ host_cuda_globals };
    auto& lt{ g.leader_thread_buffer_back };
    auto lt_stashed = lt;
    
    auto ps = from_thrust(thrust::device_vector<i2>(g.pt.p, g.pt.p + g.npt));
    auto bs = from_thrust(thrust::device_vector<i2>(g.pt.b, g.pt.b + g.npt));
    

    project_glue(1);
    ipc_pt_kernel<<<1, 1>>>(g.npt, g.pt.p, g.pt.b,
        g.cubes,
        g.lut_size, PTR(g.lut),
        PTR(g.hess.values), PTR(g.hess.inner), PTR(g.hess.outer_start),
        g.b, (float*)lt,
        nullptr, nullptr);
    CUDA_CALL(hipDeviceSynchronize());
    lt = lt_stashed;
}