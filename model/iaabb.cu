#include "hip/hip_runtime.h"

// #include "iaabb.h"
#include "cuda_header.cuh"
#include "timer.h"
#include <omp.h>
#include <ipc/distance/edge_edge.hpp>
#include <ipc/distance/point_triangle.hpp>
#include "cuda_globals.cuh"
#include <spdlog/spdlog.h>
#include <tuple>
using namespace std;
using namespace ipc;
// using namespace cuda::std;
// using namespace Eigen;
static const int max_pairs_per_thread = 512, max_aabb_list_size = 512;
// FIXME: tid probably not in 1 block

tuple<float, PointTriangleDistanceType> vf_distance(vec3f vf, Facef ff);

__device__ __host__ float vf_distance(vec3f _v, Facef f, PointTriangleDistanceType& pt_type)
{
    auto n = unit_normal(f);
    auto d = dot(n, _v - f.t0);
    auto a1 = area_x2(f.t1, f.t0, f.t2);
    auto v = _v - n * d;
    d = d * d;
    // float a2 = ((f[0] - v).cross(f[1] - v).norm() + (f[1] - v).cross(f[2] - v).norm() + (f[2] - v).cross(f[0] - v).norm());
    // auto a2 = area_x2(f[0], f[1], v) + area_x2(f[1], f[2], v) + area_x2(f[2], f[0], v);
    auto _a1 = dot(cross(f.t0 - v, f.t1 - v), n);
    auto _a2 = dot(cross(f.t1 - v, f.t2 - v), n);
    auto _a3 = dot(cross(f.t2 - v, f.t0 - v), n);
    bool inside = _a1 * _a2 > 0.0f && _a2 * _a3 > 0.0f;
    // if (a2 > a1 + 1e-8) {
    if (!inside) {
        // projection outside of triangle

        auto d_ab = h(f.t0, f.t1, v);
        auto d_bc = h(f.t1, f.t2, v);
        auto d_ac = h(f.t0, f.t2, v);

        auto d_a = ab(v, f.t0);
        auto d_b = ab(v, f.t1);
        auto d_c = ab(v, f.t2);

        auto dab = is_obtuse_triangle(f.t0, f.t1, v) ? CUDA_MIN(d_a, d_b) : d_ab;
        auto dbc = is_obtuse_triangle(f.t2, f.t1, v) ? CUDA_MIN(d_c, d_b) : d_bc;
        auto dac = is_obtuse_triangle(f.t0, f.t2, v) ? CUDA_MIN(d_a, d_c) : d_ac;

        auto d_projected = CUDA_MIN3(dab, dbc, dac);
        d += d_projected * d_projected;

        if (d_projected == d_ab)
            pt_type = PointTriangleDistanceType::P_E0;
        else if (d_projected == d_bc)
            pt_type = PointTriangleDistanceType::P_E1;
        else if (d_projected == d_ac)
            pt_type = PointTriangleDistanceType::P_E2;
        else if (d_projected == d_a)
            pt_type = PointTriangleDistanceType::P_T0;
        else if (d_projected == d_b)
            pt_type = PointTriangleDistanceType::P_T1;
        else
            pt_type = PointTriangleDistanceType::P_T2;
    }
    else
        pt_type = PointTriangleDistanceType::P_T;
    return d;
}

__global__ void aabb_intersection_test_kernel(luf* dev_aabbs, int nvi, int nfj, i2* ij, int* cnt)
{

    __shared__ luf aabbs[max_aabb_list_size];
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int n_task_per_thread = (nvi * nfj + blockDim.x - 1) / blockDim.x;
    int n_copies_per_thread = (nvi + nfj + blockDim.x - 1) / blockDim.x;
    for (int i = 0; i < n_copies_per_thread; i++) {
        int idx = tid * n_copies_per_thread + i;
        if (idx < nvi + nfj && idx < max_aabb_list_size)
            aabbs[idx] = dev_aabbs[idx];
    }
    cnt[tid] = 0;
    __syncthreads();
    // copys the bounding boxes to shared memory

    for (int _i = 0; _i < n_task_per_thread; _i++) {
        int I = tid * n_task_per_thread + _i;
        if (I < nvi * nfj) {
            int i = I / nfj;
            int j = I % nfj;
            if (intersects(aabbs[i], aabbs[nvi + j])) {
                auto put = cnt[tid]++ + tid * max_pairs_per_thread;
                ij[put] = { i, j };
            }
        }
    }
}

__global__ void inclusive_scan_kernel(int * cnt) {
    for (int i = 1; i < n_cuda_threads_per_block; i ++) {
        cnt[i] = cnt[i - 1] + cnt[i];
    }
}
__global__ void filter_distance_kernel(i2* ij, int* cnt, i2* tmp,
    // int* vilist, int* fjlist,
    vec3f* vis, Facef* fjs,
    PointTriangleDistanceType* pt_types,
    PointTriangleDistanceType* tmp_pt_types,
    float dhat = 1e-4)
{
    // // squeeze the ij list according to a prefix sum array cnt
    // // FIXME: asserting blockDim.x == n_cuda_threads_per_block
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // int start = tid == 0 ? 0 : cnt[tid - 1];
    int n_tasks = cnt[blockDim.x - 1];
    // do the vf distance and type computation
    cnt[tid] = 0;
    int n_task_per_thread = (n_tasks + blockDim.x - 1) / blockDim.x;
    for (int _i = 0; _i < n_task_per_thread; _i++) {
        int idx = tid * n_task_per_thread + _i;
        if (idx < n_tasks) {
            auto _ij = tmp[idx];
            int i = _ij[0];
            int j = _ij[1];
            // int vi = vilist[i];
            // int fj = fjlist[j];

            // compute the distance and type
            auto v{ vis[i] };
            auto f{ fjs[j] };
            auto put = cnt[tid] + tid * max_pairs_per_thread;
            auto d = vf_distance(v, f, pt_types[put]);
            if (d < dhat) {
                ij[put] = { i, j };
                cnt[tid]++;
            }
        }
    }
}
__global__ void squeeze_ij_kernel(i2* ij, int* cnt, i2* tmp, PointTriangleDistanceType* pt_types, PointTriangleDistanceType* tmp_pt_types)
{
    // squeeze again and copy back to ij matrix
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    auto start = tid == 0 ? 0 : cnt[tid - 1];
    auto copy_size = cnt[tid] - start;
    for (int i = 0; i < copy_size; i++) {
        int dst = i + start;
        int src = tid * max_pairs_per_thread + i;

        tmp[dst] = ij[src];
        tmp_pt_types[dst] = pt_types[src];
        // tmp is now dense
    }
}

void vf_col_set_cuda(
    // vector<int>& vilist, vector<int>& fjlist,
    // const std::vector<std::unique_ptr<AffineBody>>& cubes,
    // int I, int J,
    int nvi, int nfj,
    const thrust::host_vector<luf>& aabbs,
    const thrust::host_vector<vec3f>& vis,
    const thrust::host_vector<Facef>& fjs,
    const std::vector<int>& vilist, const std::vector<int>& fjlist,
    vector<array<int, 4>>& idx,
    int I, int J,
    int tid)
{
    if (nvi && nfj)
        ;
    else
        return;

#ifdef THRUST_DEV_VECTOR
    thrust::device_vector<luf> dev_aabbs(aabbs.begin(), aabbs.end());
    thrust::device_vector<vec3f> dev_vis(vis.begin(), vis.end());
    thrust::device_vector<Facef> dev_fjs(fjs.begin(), fjs.end());
    thrust::device_vector<int>
        dev_vilist(vilist.begin(), vilist.end()),
        dev_fjlist(fjlist.begin(), fjlist.end());

    // allocate memory on device
    thrust::device_vector<int> dev_cnt(n_cuda_threads_per_block, 0);
    thrust::device_vector<i2>
        ij(n_cuda_threads_per_block * max_pairs_per_thread),
        tmp(n_cuda_threads_per_block * max_pairs_per_thread);

    thrust::device_vector<PointTriangleDistanceType> pt_types(n_cuda_threads_per_block * max_pairs_per_thread, PointTriangleDistanceType::P_T), pt_types_buffer(n_cuda_threads_per_block * max_pairs_per_thread, PointTriangleDistanceType::P_T);

    auto ij_ptr = thrust::raw_pointer_cast(ij.data());
    auto cnt_ptr = thrust::raw_pointer_cast(dev_cnt.data());
    auto tmp_ptr = thrust::raw_pointer_cast(tmp.data());
    auto vilist_ptr = thrust::raw_pointer_cast(dev_vilist.data());
    auto fjlist_ptr = thrust::raw_pointer_cast(dev_fjlist.data());
    auto vis_ptr = thrust::raw_pointer_cast(dev_vis.data());
    auto fjs_ptr = thrust::raw_pointer_cast(dev_fjs.data());
    auto aabbs_ptr = thrust::raw_pointer_cast(dev_aabbs.data());
    auto pt_types_ptr = thrust::raw_pointer_cast(pt_types.data());
    auto tmp_pt_types_ptr = thrust::raw_pointer_cast(pt_types_buffer.data());
#else
    auto chunk_int = (int*)((char*)cuda_globals.buffer_chunk + tid * cuda_globals.per_stream_buffer_size);
    auto ij_size = n_cuda_threads_per_block * max_pairs_per_thread * 2;
    i2* ij_ptr = (i2*)(chunk_int);
    i2* tmp_ptr = (i2*)(chunk_int + ij_size);
    PointTriangleDistanceType* pt_types_ptr = (PointTriangleDistanceType*)(chunk_int + ij_size * 2);
    PointTriangleDistanceType* tmp_pt_types_ptr = (PointTriangleDistanceType*)(pt_types_ptr + ij_size / 2);
    int* cnt_ptr = (int*)(chunk_int + ij_size * 3);
    int *tmp_cnt = cnt_ptr + n_cuda_threads_per_block * 2;
    vec3f* vis_ptr = (vec3f*)(cnt_ptr + n_cuda_threads_per_block * 3);
    Facef* fjs_ptr = (Facef*)(vis_ptr + max_aabb_list_size);
    luf* aabbs_ptr = (luf*)(fjs_ptr + max_aabb_list_size);
    // vec3f* vis_ptr;
    // Facef* fjs_ptr;
    // luf* aabbs_ptr;
    // hipMallocManaged(&vis_ptr, vis.size() * sizeof(vec3f));
    // hipMallocManaged(&fjs_ptr, fjs.size() * sizeof(Facef));
    // hipMallocManaged(&aabbs_ptr, aabbs.size() * sizeof(luf));
    // CUDA_CALL(hipGetLastError());
    // CUDA_CALL(hipMemset(cnt_ptr, 0, n_cuda_threads_per_block));

    
    
    auto &stream {cuda_globals.streams[tid]};
    CUDA_CALL(hipMemcpyAsync((void*)vis_ptr, vis.data(), vis.size() * sizeof(vec3f), hipMemcpyHostToDevice), stream);
    CUDA_CALL(hipMemcpyAsync((void*)fjs_ptr, fjs.data(), fjs.size() * sizeof(Facef), hipMemcpyHostToDevice), stream);
    
    CUDA_CALL(hipMemcpyAsync((void*)aabbs_ptr, aabbs.data(), aabbs.size() * sizeof(luf), hipMemcpyHostToDevice), stream);
    
    // CUDA_CALL(hipMemcpy((void*)vis_ptr, vis.data(), vis.size() * sizeof(vec3f), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy((void*)fjs_ptr, fjs.data(), fjs.size() * sizeof(Facef), hipMemcpyHostToDevice));
    
    // CUDA_CALL(hipMemcpy((void*)aabbs_ptr, aabbs.data(), aabbs.size() * sizeof(luf), hipMemcpyHostToDevice));
    
    // spdlog::warn("copy complete");
#endif
    {
        // cuda kernels
        aabb_intersection_test_kernel<<<1, n_cuda_threads_per_block, 0, stream>>>(aabbs_ptr, nvi, nfj, ij_ptr, cnt_ptr);

        CUDA_CALL(hipGetLastError());

        // thrust::inclusive_scan(thrust::cuda::par_nosync, dev_cnt.begin(), dev_cnt.end(), dev_cnt.begin());
        // thrust::inclusive_scan(thrust::cuda::par_nosync, cnt_ptr, cnt_ptr + n_cuda_threads_per_block, cnt_ptr);
        inclusive_scan_kernel<<<1, 1, 0, stream>>>(cnt_ptr);
        CUDA_CALL(hipGetLastError());

        squeeze_ij_kernel<<<1, n_cuda_threads_per_block, 0, stream>>>(ij_ptr, cnt_ptr, tmp_ptr, pt_types_ptr, tmp_pt_types_ptr);

        // now culled pairs are gathered in the front of tmp
        // dev_cnt.back() has the length of the culled pairs

        bool run_on_gpu = true, run_thrust = false, run_kernel = !run_thrust;
        if (run_on_gpu) {
            if (run_kernel) {
                // pass
                filter_distance_kernel<<<1, n_cuda_threads_per_block, 0, stream>>>(ij_ptr, cnt_ptr, tmp_ptr,
                    // vilist_ptr, fjlist_ptr,
                    vis_ptr, fjs_ptr, pt_types_ptr, tmp_pt_types_ptr);

                CUDA_CALL(hipGetLastError());
                inclusive_scan_kernel<<<1, 1, 0, stream>>>(cnt_ptr);
                // thrust::inclusive_scan(thrust::cuda::par_nosync, cnt_ptr, cnt_ptr + n_cuda_threads_per_block, cnt_ptr);

                // thrust::inclusive_scan(thrust::cuda::par_nosync, dev_cnt.begin(), dev_cnt.end(), dev_cnt.begin());

                squeeze_ij_kernel<<<1, n_cuda_threads_per_block, 0, stream>>>(ij_ptr, cnt_ptr, tmp_ptr, pt_types_ptr, tmp_pt_types_ptr);
                CUDA_CALL(hipStreamSynchronize(stream));
                
            }
            else {
#ifdef THRUST_DEV_VECTOR
                // pass
                int cnt_gpu = dev_cnt.back();
                auto ij_end = thrust::copy_if(thrust::device, tmp.begin(), tmp.begin() + cnt_gpu, ij.begin(), [=] __device__(i2 I) -> bool {
                    auto i = I[0], j = I[1];
                    PointTriangleDistanceType ptt;
                    auto vi{ vis_ptr[i] };
                    auto fj{ fjs_ptr[j] };
                    auto d = vf_distance(vis_ptr[i], fjs_ptr[j], ptt);
                    // __device__ float d = 1e-5f;
                    return (d < 1e-4f);
                });
                thrust::copy(thrust::device, ij.begin(), ij_end, tmp.begin());
                dev_cnt.back() = ij_end - ij.begin();
#endif
            }
        }
        else {
#ifdef THRUST_DEV_VECTOR
            // run_on_gpu = false, pass
            // NOTE: copy_if does not relocate the space. should allocate storage manually
            if (run_thrust) {
                int n_copy = dev_cnt.back();
                thrust::host_vector<i2> host_ij(tmp.begin(), tmp.begin() + n_copy), host_tmp(n_copy);
                int cnt = 0;
                thrust::copy_if(thrust::host, host_ij.begin(), host_ij.begin() + n_copy, host_tmp.begin(), [&](i2 I) -> bool {
                    auto i = I[0], j = I[1];
                    PointTriangleDistanceType ptt;
                    auto vi{ vis[i] };
                    auto fj{ fjs[j] };
                    auto d = vf_distance(vi, fj, ptt);
                    // __device__ float d = 1e-5f;
                    bool ret = d < 1e-4f;
                    if (ret) cnt++;
                    return ret;
                });
                // thrust::inclusive_scan(thrust::host, dev_cnt.begin(), dev_cnt.end(), dev_cnt.begin());
                thrust::copy(host_tmp.begin(), host_tmp.begin() + cnt, tmp.begin());
                dev_cnt.back() = cnt;
            }
            else {
                // exact solution on cpu, pass
                thrust::host_vector<i2> host_ij;

                for (int i = 0; i < nvi; i++)
                    for (int j = 0; j < nfj; j++)
                        if (intersects(aabbs[i], aabbs[j + nvi])) {
                            PointTriangleDistanceType ptt;
                            auto d = vf_distance(vis[i], fjs[j], ptt);
                            auto tup = vf_distance(vis[i], fjs[j]);
                            auto d2 = std::get<0>(tup);
                            auto type_ref = std::get<1>(tup);
                            if (d < 1e-4f) {
                                host_ij.push_back({ i, j });
                            }
                            if (fabs(d - d2) > 1e-6f) {
                                spdlog::error("d1 = {}, d2 = {}", d, d2);
                                spdlog::error("type, cuda = {}, ref = {}", static_cast<cuda::std::underlying_type_t<ipc::PointTriangleDistanceType>>(ptt), static_cast<cuda::std::underlying_type_t<ipc::PointTriangleDistanceType>>(type_ref));
                            }
                        }
                thrust::copy(host_ij.begin(), host_ij.end(), tmp.begin());
                dev_cnt.back() = host_ij.size();
            }
#endif
        }
    }
    // now tmp has the exact collsion set with d < dhat,
    // and dev_cnt.back has the information of the length of the set

    {
        int n_collision_set = cnt_ptr[n_cuda_threads_per_block - 1];
        thrust::host_vector<i2> host_ij(tmp_ptr, tmp_ptr + n_collision_set);
        for (int i = 0; i < n_collision_set; i++) {
            auto vi = host_ij[i][0], fj = host_ij[i][1];
            
            idx.push_back({ I, vilist[vi], J, fjlist[fj] });
        }
    }
}

#include "cuda_globals.cuh"
#include <assert.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <cuda/std/tuple>
#include <cuda/std/type_traits>
#include <thrust/set_operations.h>
void stencil_classifier(
    thrust::device_vector<i2>& pt_idx,
    thrust::device_vector<i2>& pt_body_idx,
    thrust::device_vector<PointTriangleDistanceType>& pt_types)
{
    static const PointTriangleDistanceType types[] = {
        PointTriangleDistanceType::P_T0, // 0
        PointTriangleDistanceType::P_T1, // 1
        PointTriangleDistanceType::P_T2, // 2
        PointTriangleDistanceType::P_E0, // 3
        PointTriangleDistanceType::P_E1, // 4
        PointTriangleDistanceType::P_E2, // 5
        PointTriangleDistanceType::P_T // 6
    };
    // static const EdgeEdgeDistanceType edge_types[] = {
    // };
    assert(pt_idx.size() == pt_types.size() && pt_idx.size() == pt_body_idx.size());
    for (int i = 0; i < 7; i++) {
        auto &cset{ cuda_globals.collision_sets.pt_set[i] }, &bset{ cuda_globals.collision_sets.pt_set_body_index[i] };
        // cset.clear();
        // bset.clear();
        thrust::copy_if(
            thrust::make_zip_iterator(thrust::make_tuple(pt_idx.begin(), pt_body_idx.begin(), pt_types.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(pt_idx.end(), pt_body_idx.end(), pt_types.end())),
            thrust::make_zip_iterator(thrust::make_tuple(cset.end(), bset.end(), thrust::make_discard_iterator())),
            [=] __device__(const thrust::tuple<i2, i2, PointTriangleDistanceType>& tup) {
                return static_cast<cuda::std::underlying_type_t<PointTriangleDistanceType>>(thrust::get<2>(tup)) == i;
            });
    }
    // FIXME: make sure i < j before merging into lut? 
    {
    // static thrust::device_vector<i2> merged_lut;
        // // generate look up table for spare hess 
        // merged_lut .resize(0);
        // thrust::set_union(thrust::device, pt_body_idx.begin(), pt_body_idx.end(), cuda_globals.lut.begin(), cuda_globals.lut.end(), merged_lut.begin());
        // cuda_globals.lut = merged_lut;

        // should be sorted, just not worth it

        thrust::copy(pt_body_idx.begin(), pt_body_idx.end(), cuda_globals.lut.end());
    }
}

CudaGlobals::CudaGlobals()
{
    hipGetDevice(&device_id);
    int n_proc = omp_get_num_procs();
    per_stream_buffer_size = n_cuda_threads_per_block * max_pairs_per_thread * sizeof(i2) * 4;
    hipMallocManaged(&buffer_chunk, per_stream_buffer_size * n_proc);
    streams = new hipStream_t[n_proc];
    for (int i = 0; i < n_proc; i++) {
        hipStreamCreate(&streams[i]);
    }
    
}

CudaGlobals::~CudaGlobals()
{
    hipFree(buffer_chunk);
    for (int i = 0 ; i < omp_get_num_procs(); i++) {
        hipStreamDestroy(streams[i]);
    }
    delete[] streams;
}
