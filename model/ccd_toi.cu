#include "cuda_header.cuh"



inline __host__ __device__ vec3f linerp(vec3f pt1, vec3f pt0, float t){
    return t * pt1 + (1.0f - t) * pt0;
}
inline __host__ __device__ Facef linerp(Facef tt1, Facef tt0, float t) {
    return Facef {
        linerp(tt1.t0, tt0.t0, t),
        linerp(tt1.t1, tt0.t1, t),
        linerp(tt1.t2, tt0.t2, t),
    };
}
inline __host__ __device__ Edgef linerp(Edgef et1, Edgef et0, float t) {
    return Edgef {
        linerp(et1.e0, et0.e0, t),
        linerp(et1.e1, et0.e1, t),
    };
}

__host__ __device__ void cubic_binomial (float a[3], float b[3], float ret_polynomial[4]){
    polynomial[0] += b[0] * b[1] * b[2];
    polynomial[1] += a[0] * b[1] * b[2] + b[0] * b[1] * a[2] + b[0] * a[1] * b[2];
    polynomial[2] += a[0] * a[1] * b[2] + b[0] * a[1] * a[2] + a[0] * b[1] * a[2];
    polynomial[3] += a[0] * a[1] * a[2];
}

__forceinline__ int rc_3(int i, int j) {
    return 3 * j + i;
}
__host__ __device__ void det_polynomial(
    // const mat3& a, const mat3& b
    float *a, float *b, float *ret_polynomial
)
{
    float pos_polynomial[4]{ 0.0 }, neg_polynomial[4]{ 0.0 };
    float c11c22c33[2][3]{
        { a[rc_3(0, 0)], a[rc_3(1, 1)], a[rc_3(2, 2)] },
        { b[rc_3(0, 0)], b[rc_3(1, 1)], b[rc_3(2, 2)] }
    },
        c12c23c31[2][3]{
            { a[rc_3(0, 1)], a[rc_3(1, 2)], a[rc_3(2, 0)] },
            { b[rc_3(0, 1)], b[rc_3(1, 2)], b[rc_3(2, 0)] }
        },
        c13c21c32[2][3]{
            { a[rc_3(0, 2)], a[rc_3(1, 0)], a[rc_3(2, 1)] },
            { b[rc_3(0, 2)], b[rc_3(1, 0)], b[rc_3(2, 1)] }
        };
    float c11c23c32[2][3]{
        { a[rc_3(0, 0)], a[rc_3(1, 2)], a[rc_3(2, 1)] }, { b[rc_3(0, 0)], b[rc_3(1, 2)], b[rc_3(2, 1)] }
    },
        c12c21c33[2][3]{
            { a[rc_3(0, 1)], a[rc_3(1, 0)], a[rc_3(2, 2)] }, { b[rc_3(0, 1)], b[rc_3(1, 0)], b[rc_3(2, 2)] }
        },
        c13c22c31[2][3]{
            { a[rc_3(0, 2)], a[rc_3(1, 1)], a[rc_3(2, 0)] }, { b[rc_3(0, 2)], b[rc_3(1, 1)], b[rc_3(2, 0)] }
        };
    cubic_binomial(
        c11c22c33[0],
        c11c22c33[1],
        pos_polynomial);
    cubic_binomial(
        c12c23c31[0],
        c12c23c31[1],
        pos_polynomial);
    cubic_binomial(
        c13c21c32[0],
        c13c21c32[1],
        pos_polynomial);
    cubic_binomial(
        c11c23c32[0],
        c11c23c32[1],
        neg_polynomial);
    cubic_binomial(
        c12c21c33[0],
        c12c21c33[1],
        neg_polynomial);
    cubic_binomial(
        c13c22c31[0],
        c13c22c31[1],
        neg_polynomial);
    for (int i = 0; i < 4; i++) ret_polynomial[i] = pos_polynomial[i] - neg_polynomial[i];
}



int build_and_solve_4_points_coplanar(
    const vec3f& p0_t0,
    const vec3f& p1_t0,
    const vec3f& p2_t0,
    const vec3f& p3_t0,

    const vec3f& p0_t1,
    const vec3f& p1_t1,
    const vec3f& p2_t1,
    const vec3f& p3_t1,

    float roots[3])
{

    float a1[9] {
        pt_t1.x - p1_t0.x, pt_t1.y - p1_t0.y, pt_t1.z - p1_t0.z,
        pt_t2.x - p2_t0.x, pt_t2.y - p2_t0.y, pt_t2.z - p2_t0.z,
        pt_t3.x - p3_t0.x, pt_t3.y - p3_t0.y, pt_t3.z - p3_t0.z
    },
    a2[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p2_t1.x - p2_t0.x, p2_t1.y - p2_t0.y, p2_t1.z - p2_t0.z,
        p3_t1.x - p3_t0.x, p3_t1.y - p3_t0.y, p3_t1.z - p3_t0.z
    }, 
    a3[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p1_t1.x - p1_t0.x, p1_t1.y - p1_t0.y, p1_t1.z - p1_t0.z,
        p3_t1.x - p3_t0.x, p3_t1.y - p3_t0.y, p3_t1.z - p3_t0.z
    },
    a4[9] {
        p0_t1.x - p0_t0.x, p0_t1.y - p0_t0.y, p0_t1.z - p0_t0.z,
        p1_t1.x - p1_t0.x, p1_t1.y - p1_t0.y, p1_t1.z - p1_t0.z,
        p2_t1.x - p2_t0.x, p2_t1.y - p2_t0.y, p2_t1.z - p2_t0.z
    };
    float b1[9] {
        p1_t0.x, p1_t0.y, p1_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    }, 
    b2[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    },
    b3[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p1_t0.x, p1_t0.y, p1_t0.z,
        p3_t0.x, p3_t0.y, p3_t0.z
    },
    b4[9] {
        p0_t0.x, p0_t0.y, p0_t0.z,
        p1_t0.x, p1_t0.y, p1_t0.z,
        p2_t0.x, p2_t0.y, p2_t0.z
    }; 
    
    float ret_polynomial[4] {0.0f};
    float tmp_polynomial[4] {0.0f};

    det_polynomial(a1, b1, ret_polynomial);
    det_polynomial(a2, b2, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] -= tmp_polynomial[i];
    det_polynomial(a3, b3, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] += tmp_polynomial[i];
    det_polynomial(a4, b4, tmp_polynomial);
    for (int i = 0 ; i < 4; i ++) ret_polynomial[i] -= tmp_polynomial[i];

    double root = 1.0;
    int found = cubic_roots(roots, ret_polynomial, 0.0, 1.0);
    return found;
}


__device__ __host__ bool _cross(const Edgef &ei, const Edgef &ej){
    auto vei = ei.e1 - ei.e0;
    auto vej0 = ej.e0 - ei.e0;
    auto vej1 = ej.e1 - ei.e0;
    return (cross(vei, vej0), cross(vei, vej1)) < 0.0f;
}

__device__ __host__ bool verify_root_ee(
    const Edgef &ei, 
    const Edgef &ej
) {
    return _cross(ei, ej) && _cross(ej, ei);
}

__forceinline__ __device__ __host__ bool inside(const Facef &f, const vec3f &p) {
    auto f01 = cross(t0 - p, t1- p);
    auto f12 = cross(t1 - p, t2- p);
    auto f20 = cross(t2 - p, t0- p);
    return dot(f01, f12) >= 0.0f && dot(f12, f20) >= 0.0f;
}

__device__ __host__ verify_root_pt(
    const vec3f &p, const Facef &f
 ) {
    auto n = f.unit_normal();
    double d = dot(n, p - f.t0);
    auto v = p - d * n;
    return inside(f, v);
 }
__device__ __host__ float pt_collision_time(
    const vec3f &p0,
    const Facef &t0, 
    const vec3f &p1,
    const Facef &t1
){
    float roots[3];
    int found = build_and_solve_4_points_coplanar(p0, t0.t0, t0.t1, t0.t2, p1, t1.t0, t1.t1, t1.t2, roots);
    bool true_root = false;
    for (int i = 0; i < found && !true_root; i ++) {
        root = roots[i];
        true_root = verify_root_pt(linerp(p1, p0, root), linerp(t1, t0, root));
    }
    return found && true_root? root: 1.0f;
}

__device__ __host__ float ee_collision_time(
    const Edgef &ei0, 
    const Edgef &ej0,
    const Edgef &ei1,
    const Edgef &ej1
){
    float roots[3];
    int found = build_and_solve_4_points_coplanar(
        ei0.e0, ei0.e1, ej0.e0, ej0.e1,
        ei1.e0, ei1.e1, ej1.e0, ej1.e1,
        roots
    );
    float root = 1.0f;
    bool true_root = false;
    for (int i = 0; i < found && !true_root; i ++) {
        root = roots[i];
        true_root = verify_root_ee(linerp(ei1, ei0, root), lerp(ej1, ej0, root));
    }
    return found && true_root? root: 1.0f;
}